
#include "hip/hip_runtime.h"


#include <stdio.h>

//////////////////////////////
// Function pointer types ////
//////////////////////////////

typedef void(*InitFunction)(float &);
typedef float(*OperationType1)(float, float);
typedef float(*OperationType2)(float, float&);

////////////////////////////////////////////////
// The Kernel we're sending the function ptrs //
////////////////////////////////////////////////
__global__ void ExecuteModelKernel(float *inA, float *inB, float *out, int N,
	InitFunction init,
	OperationType1 op1,
	OperationType2 op2)
{
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	if (id < N)
	{
		// Call init function
		if (init != NULL)
		{
			init(out[id]);
			inA[id] = 1.0f;
			inB[id] = 2.0f;
		}
			

		// Operation 1
		if (op1 != NULL)
		{
			out[id] = op1(inA[id], inB[id]);
		}

		// Operation 2
		if (op2 != NULL)
		{
			out[id] = op2(inA[id], inB[id]);
		}
			

		printf("%d: %f\n", id, out[id]);
	}
}

/////////////////////////////////
// The functions to point to ////
/////////////////////////////////

// InitFunction methods 
__device__
void Zero(float &a)
{
	a = 0;
}

__device__
void Unity(float &a)
{
	a = 1;
}
__device__
void Ignore(float &a)
{
}

__device__ InitFunction dZero = Zero;
__device__ InitFunction dUnity = Unity;
__device__ InitFunction dIgnore = Ignore;

// OperationType1 methods 
__device__
float Add(float _a, float _b)
{
	return _a + _b;
}

__device__
float Subtract(float a, float b)
{
	return a - b;
}

__device__
float Times(float a, float b)
{
	return a * b;
}

__device__ OperationType1 dAdd = Add;
__device__ OperationType1 dSubtract = Subtract;
__device__ OperationType1 dTimes = Times;

// OperationType2 methods
__device__
float AddAndTimes(float a, float &b)
{
	float result = a + b;
	b *= a;
	return result;
}

__device__
float SubtractAndDivide(float a, float &b)
{
	float result = a - b;
	b = a / b;
	return result;
}

__device__
float TimesAndDivide(float a, float &b)
{
	float result = a * b;
	b = a / b;
	return result;
}

__device__ OperationType2 dAddAndTimes = AddAndTimes;
__device__ OperationType2 dSubtractAndDivide = SubtractAndDivide;
__device__ OperationType2 dTimesAndDivide = TimesAndDivide;

///////////////////////////////////////////////////
// A Model that stores pointers to these methods //
// do define runtime behavior with a model ////////
///////////////////////////////////////////////////

// Abstract parent
// lives on host always
class BaseFunctionModel
{
public:
	InitFunction Init;
	OperationType1 Op1;
	OperationType2 Op2;

	//__host__ __device__
	//virtual ~BaseFunctionModel() = 0;
};

// A specific model 
class FunctionModelA : public BaseFunctionModel
{
public:
	//__host__ __device__
	FunctionModelA()
	{
		// assign the functions that define 
		// the model in the constructor
		hipMemcpyFromSymbol(&Init, HIP_SYMBOL(dUnity), sizeof(InitFunction));
		hipMemcpyFromSymbol(&Op1, HIP_SYMBOL(dAdd), sizeof(OperationType1));
		hipMemcpyFromSymbol(&Op2, HIP_SYMBOL(dTimesAndDivide), sizeof(OperationType2));
	}

	//__host__ __device__
	~FunctionModelA()
	{
		Init = NULL;
		Op1 = NULL;
		Op2 = NULL;
	}
};

// Another different specific model 
class FunctionModelB : public BaseFunctionModel
{
public:
	//__host__ __device__
	FunctionModelB()
	{
		// assign the functions that define 
		// the model in the constructor
		hipMemcpyFromSymbol(&Init, HIP_SYMBOL(dUnity), sizeof(InitFunction));
		hipMemcpyFromSymbol(&Op1, HIP_SYMBOL(dSubtract), sizeof(OperationType1));
		hipMemcpyFromSymbol(&Op2, HIP_SYMBOL(dSubtractAndDivide), sizeof(OperationType2));
	}

	//__host__ __device__
	~FunctionModelB()
	{
		Init = NULL;
		Op1 = NULL;
		Op2 = NULL;
	}
};

///////////////////////////////////////
// The Data Model that lives on host //
// and wraps function pointers       //
///////////////////////////////////////

class DataModel
{
public:
	int N;
	float *a;
	float *b;
	float *c;

	//__host__
	DataModel(int N)
	{
		this->N = N;
	}

	//__host__
	void AllocGpu()
	{
		size_t size = sizeof(float) * N;
		hipMalloc((void**)&a, size);
		hipMalloc((void**)&b, size);
		hipMalloc((void**)&c, size);
		hipMemset(a, 1, size);
		hipMemset(b, 1, size);
		hipMemset(c, 1, size);
	}
};



hipError_t ExecuteModel(DataModel *data, BaseFunctionModel *model);

int main()
{
	int L = 16;
	 
	// get the data on the gpu with data model pointing to it
	DataModel *dataA = new DataModel(L);
	DataModel *dataB = new DataModel(L);
	dataA->AllocGpu();
	dataB->AllocGpu();

	// define the behavior you'd like
	BaseFunctionModel *modelA = new FunctionModelA();
	BaseFunctionModel *modelB = new FunctionModelB();

	// execute with same kernel but receie different results
	hipError_t errA = ExecuteModel(dataA, modelA);
	hipError_t errB = ExecuteModel(dataB, modelB);

	printf("%s: %s\n%s: %s\n", hipGetErrorName(errA)
		,hipGetErrorString(errA) 
		,hipGetErrorName(errB)
		,hipGetErrorString(errB)
	);

	// copy the data back
	size_t size = sizeof(float) * L;
	float *aA = new float[L];
	float *bA = new float[L];
	float *cA = new float[L];

	hipMemcpy(aA, dataA->a, size, hipMemcpyDeviceToHost);
	hipMemcpy(bA, dataA->b, size, hipMemcpyDeviceToHost);
	hipMemcpy(cA, dataA->c, size, hipMemcpyDeviceToHost);

	// copy the data back
	float *aB = new float[L];
	float *bB = new float[L];
	float *cB = new float[L];
	hipMemcpy(aB, dataB->a, size, hipMemcpyDeviceToHost);
	hipMemcpy(bB, dataB->b, size, hipMemcpyDeviceToHost);
	hipMemcpy(cB, dataB->c, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < L; i++)
	{
		printf("A: %f %f %f \t B: %f %f %f\n", aA[i], bA[i], cA[i], aB[i], bB[i], cB[i]);
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t ExecuteModel(DataModel *data, BaseFunctionModel *model)
{
	ExecuteModelKernel <<< 4, data->N / 4 >>>
		(
			data->a,
			data->b,
			data->c,
			data->N,
			model->Init,
			model->Op1,
			NULL
			);

	return hipGetLastError();
}
